#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo method to calculate PI
// Inefficient code using CURAND
// solution 2020
// execute using nvcc MCPi2020.cu -I /usr/local/cuda-10.0/samples/common/inc -lcurand
////////////////////////////////////////////////////////////////////////

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <ctime>

using namespace std;


__global__ void calcPI (int n, float *d_x, float *d_y, int *d_count, int iter )
// Applies Monte Carlo method to compute PI 
{
   int i = (blockIdx.x*blockDim.x + threadIdx.x);
   int index = i * iter;  // as each processor does a number of iterations
   double x,y,z;
   int tempcount = d_count[i];
   if (i<n)   // to prevent excess processors from accessing out-of-bounds data
   { for(int j=0; j<iter; j++)   // iterations of Monte Carlo - based on parameter iter
      { x = d_x[index+j];   // optimise using only one array ....
        y = d_y[index+j]; 
        z = x*x+y*y;
        if (z<=1) tempcount++;   
      }
    d_count[i] = tempcount;
   }
}


int main (int argc, char*argv[] )
{		int iterations = 100; // number of iterations for MC alg per processor
		int n = 1000000; // total no. of iterations of MC // max storage = 1 500 000
		int block = 32; // block size
		int grid = n / block + 1;  // round up number of blocks

		int count=0; /* # of points in the 1st quadrant of unit circle */
		double pi;


 // initialise card

	findCudaDevice(argc, (const char**) argv);   // from hip/hip_runtime_api.h 
 
  // initialise CUDA timing

	float milli;
	hipEvent_t start, stop, starttot, stoptot;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	clock_t iStart, iEnd;
	iStart = clock();

	// allocate memory on host and device
		int *h_count = (int *)malloc(sizeof(int)*n);    // for processor sums
		float *d_x;  // memory for initial x random numbers
		checkCudaErrors(hipMalloc((void**)&d_x,sizeof(float)*n*iterations));
		float *d_y;  // memory for initial y random numbers
		checkCudaErrors(hipMalloc((void**)&d_y,sizeof(float)*n*iterations));
		int *d_count; // memory for intermediate results
		checkCudaErrors(hipMalloc((void**)&d_count,sizeof(int)*n));
 		checkCudaErrors(hipMemset(d_count,0, sizeof(int)*n));
 
	// random number generation
		hipEventRecord(start);  // start timing

		hiprandGenerator_t gen;
		checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
		checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
		checkCudaErrors( hiprandGenerateUniform(gen, d_x, n*iterations) );
 		checkCudaErrors( hiprandGenerateUniform(gen, d_y, n*iterations) );
 
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);  // time random generation

		printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
          milli, n/(0.001*milli));

	// execute kernel and time it

		hipEventRecord(start); // start timing

		calcPI<<<grid,block>>>(n, d_x, d_y, d_count, iterations);

		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);  // stop timing actual kernel execution

		printf("Square root kernel execution time (ms): %f \n",milli);

		checkCudaErrors(hipDeviceSynchronize());  // flush print queues

		// copy back results
		checkCudaErrors( hipMemcpy(h_count, d_count, sizeof(int)*n, hipMemcpyDeviceToHost) );

		for (int k = 0; k < n; k++) // sum all counts
		 {  count += h_count[k];
		    // printf("count %i = %i \n", k, h_count[k]); 
		  }  

		pi=(double)count/(n*iterations)*4;

		iEnd = clock();

		printf("Total PI execution time (ms): %g \n", (double) ((iEnd-iStart) * 1000 / CLOCKS_PER_SEC)); 

		printf("# of trials= %d , estimate of pi is %g \n",n*iterations,pi);
	
	 // Tidy up library

		checkCudaErrors( hiprandDestroyGenerator(gen) );

	// Release memory and exit cleanly

		free(h_count);
		checkCudaErrors( hipFree(d_count) );
		checkCudaErrors( hipFree(d_x) );
		checkCudaErrors( hipFree(d_y) );

	// CUDA exit 

		hipDeviceReset();
}


